#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include "hip/hip_runtime.h"
#include <iostream>
#include ""
#include "hip/device_functions.h"
#include <stdio.h>
#include "constsmacros.h"
#include <stdlib.h>
#include <math.h>
#include "ImageLoading.cu"
//#include "CUDAArray.cuh"
#include <float.h>
#include "OrientationField.cu"
#include "Convolution.cu"

#define M_PI 3.14159265358979323846
#define Pi4 (M_PI / 4);

class RidgeOnLine
{
	
private:
	int _step;
	int _sizeSection;
	int _height;
	int _width;

public:
	RidgeOnLine()
	{
		
	}

	~RidgeOnLine()
	{

	}
};

__device__ __host__
Point NewPoint(int x, int y)
{
	Point newP;
	newP.x = x;
	newP.y = y;
	return newP;
}

__device__ void AddMinutiae(CUDAArray<int>* countOfMinutiae, CUDAArray<Minutiae>* minutiaes, Minutiae minutiae, int* indexOfMinutiae)
{
	//minutiaes.At(blockIdx.x * gridDim.x + blockIdx.y, 0)->Add(minutiae);
	//printf("Adding a minutiae\n");
	minutiaes->SetAt(0, *indexOfMinutiae, minutiae);
	int past = countOfMinutiae->At(0, blockIdx.x * gridDim.x + blockIdx.y);
	countOfMinutiae->SetAt(0, blockIdx.x * gridDim.x + blockIdx.y, past + 1);
	*countOfMinutiae++;
	//printf("%d %d: Was %d. Must be %d, but in real %d\n", blockIdx.x, blockIdx.y, past, past + 1, countOfMinutiae->At(blockIdx.x * gridDim.x + blockIdx.y, 0));
}

__device__ bool OutOfImage(CUDAArray<float> image, int x, int y, int partX, int partY)
{
	//return (x < 0) || (y < 0) || (y >= image.Width) || (x >= image.Height);
	return (x < blockIdx.x * partX) || (y < blockIdx.y * partY) || (x >= (blockIdx.x + 1) * partX) || (y >= (blockIdx.y + 1) * partY) || (y >= image.Width) || (x >= image.Height);
}

__device__ void NewSection(int sx, int sy, Direction direction, CUDAArray<float> image, CUDAArray<float> orientationField, 
	Point* section, float* sectionAngle, int* centerSection, bool* flag, int size, int partX, int partY)
{
	int lengthWings = size / 2;

	for (int i = 0; i < size; i++)
	{
		section[i] = NewPoint(-1, -1);

	}

	int x = sx;
	int y = sy;

	int lEnd = lengthWings;
	int rEnd = lEnd;

	bool rightE = false;
	bool leftE = false;

	float angle = orientationField.At(x, y) + M_PI / 2;

	for (int i = 1; i <= lengthWings; i++)
	{
		int xs = (int)(x - i * cos(angle));
		int ys = (int)(y - i * sin(angle));
		int xe = (int)(x + i * cos(angle));
		int ye = (int)(y + i * sin(angle));

		if (!OutOfImage(image, xs, ys, partX, partY) && (image.At(xs, ys) < 15) && !rightE)
		{
			section[lengthWings - i] = NewPoint(xs, ys);
			rEnd--;
		}
		else
		{
			rightE = true;
		}

		if (!OutOfImage(image, xe, ye, partX, partY) && (image.At(xe, ye) < 15) && !leftE)
		{
			section[lengthWings - i] = NewPoint(xe, ye);
			lEnd--;
		}
		else
		{
			leftE = true;
		}

		*centerSection = (lEnd + rEnd) / 2;

		x = section[*centerSection].x;
		y = section[*centerSection].y;
	}

	angle = orientationField.At(x, y) + direction * M_PI;
	if (angle < 0) angle += 2 * M_PI;

	if (*flag){
		if (abs(*sectionAngle - angle) > 0.2 && abs(*sectionAngle - angle) < 6) angle + M_PI;
	}
	else *flag = true;

	*sectionAngle = angle;
}



__device__ bool CheckAndDeleteFalseMinutia(Minutiae minutia)
{
	return false;
}

__device__ Point MakeStep(CUDAArray<float> image, Point* section, int* centerSection, float* sectionAngle, int step, int partX, int partY)
{
	int x = section[*centerSection].x;
	int y = section[*centerSection].y;

	float dx = (float)x + (float)step * cos(*sectionAngle);
	float dy = (float)y + (float)step * sin(*sectionAngle);

	x = (int)(dx >= 0 ? dx + 0.5 : dx - 0.5);
	y = (int)(dy >= 0 ? dy + 0.5 : dy - 0.5);

	return OutOfImage(image, x, y, partX, partY) ? NewPoint(-1, -1) : NewPoint(x, y);
}

__device__ MinutiaeType CheckStopCriteria(CUDAArray<float> image, CUDAArray<bool> visited, Point* section, int* centerSection, int threshold = 20)
{
	int x = section[*centerSection].x;
	int y = section[*centerSection].y;

	if (visited.At(x, y))
		return Intersection;
	if (image.At(x, y) > threshold)
		return LineEnding;

	return NotMinutiae;
}

__device__ Point queue[30 * 15 * 15];

__device__ void Paint(CUDAArray<float> image, CUDAArray<bool> visited, Point* oldSection, Point* section, int size, int partX, int partY)
{
	//Queue* queue = new Queue;
	int shift = 30 * (blockIdx.x * gridDim.x + blockIdx.y);
	int rPointer = 0, lPointer = 0;
	//queue = (Point*)malloc(32 * 32 * sizeof(Point));
	Point v1, v2;

	int x1 = -1, x2 = -1, y1 = -1, y2 = -1, x_a, y_a;

	for (int i = 0; i < size; i++)
	{
		if (oldSection[i].x == -1) continue;

		if (x1 == -1)
		{
			x1 = oldSection[i].x;
			y1 = oldSection[i].y;
		}

		x2 = oldSection[i].x;
		y2 = oldSection[i].y;

		visited.SetAt(oldSection[i].x, oldSection[i].y, true);
		queue[shift + rPointer] = oldSection[i];
		rPointer++;
	}

	v1 = NewPoint(x2 - x1, y2 - y1);
	x_a = x1;
	y_a = y1;

	x1 = -1;
	y1 = -1;
	x2 = -1;
	y2 = -1;

	for (int i = 0; i < size; i++)
	{
		if (section[i].x == -1) continue;

		if (x1 == -1)
		{
			x1 = section[i].x;
			y1 = section[i].y;
		}

		x2 = section[i].x;
		y2 = section[i].y;

		visited.SetAt(section[i].x, section[i].y, true);
		//printf("%d %d: Point (%d, %d) has been choosed as stop point\n", blockIdx.x, blockIdx.y, i, j);
		queue[shift + rPointer] = section[i];
		rPointer++;
	}

	v2 = NewPoint(x2 - x1, y2 - y1);

	if (v1.x*v2.x + v1.y*v2.y < 0)
	{
		x1 = x2;
		y1 = y2;
		v1 = NewPoint(-v1.x, -v1.y);
	}

	while (rPointer - lPointer > 0)
	{
		Point point = queue[shift + lPointer];
		lPointer++;

		int cX = point.x;
		int cY = point.y;

		for (int i = -1; i < 2; i++)
			for (int j = -1; j < 2; j++)
			{
				if (i == 0 && j == 0) continue;

				int x = cX + i;
				int y = cY + j;

				if (OutOfImage(image, x, y, partX, partY) || visited.At(x, y) || image.At(x, y) > 15) continue;

				Point pointV1 = NewPoint(x_a - x, y_a - y);
				Point pointV2 = NewPoint(x1 - x, y1 - y);

				int skew1 = v1.x*pointV1.y - pointV1.x*v1.y >= 0 ? 1 : -1;
				int skew2 = v2.x*pointV2.y - pointV2.x*v2.y >= 0 ? 1 : -1;

				if (skew1*skew2 < 0)
				{
					queue[shift + rPointer] = NewPoint(x, y);
					rPointer++;
					visited.SetAt(x, y, true);
					//printf("%d %d: Point (%d, %d) has been visited\n", blockIdx.x, blockIdx.y, x, y);
				}
			}
	}
}

__device__ void FollowLine(int x, int y, Direction direction, CUDAArray<float> image, CUDAArray<float> orientationField,
	CUDAArray<bool> visited, CUDAArray<int> countOfMinutiae, CUDAArray<Minutiae> minutiaes,
	Point* section, float* sectionAngle, int* centerSection, bool* flag, int size, int step, int partX, int partY, int* indexOfMinutiae)
{
	NewSection(x, y, direction, image, orientationField, section, sectionAngle, centerSection, flag, size, partX, partY);
	if (section[*centerSection].x == -1) return;

	MinutiaeType type;
	Point point;

	do
	{
		Point* oldSection = new Point[size];
		for (int i = 0; i < size; i++)
			oldSection[i] = section[i];

		point = MakeStep(image, section, centerSection, sectionAngle, step, partX, partY);

		if (point.x == -1) return;

		NewSection(point.x, point.y, direction, image, orientationField, section, sectionAngle, centerSection, flag, size, partX, partY);
		if (section[*centerSection].x == -1) return;

		type = CheckStopCriteria(image, visited, section, centerSection);

		Paint(image, visited, oldSection, section, size, partX, partY);
	} while (type == NotMinutiae);

	Minutiae possMinutiae;
	possMinutiae.x = point.x;
	possMinutiae.y = point.y;
	possMinutiae.angle = *sectionAngle;
	possMinutiae.type = type;

	//printf("Minutia. x = %d y = %d type = %d\n", possMinutiae.x, possMinutiae.y, possMinutiae.type);

	//if (IsDuplicate(possMinutiae)) return;

	if (!CheckAndDeleteFalseMinutia(possMinutiae))
	{
		//printf("Minutia. x = %d y = %d type = %d\n", possMinutiae.x, possMinutiae.y, possMinutiae.type);
		AddMinutiae(&countOfMinutiae, &minutiaes, possMinutiae, indexOfMinutiae);
	}
}

__global__ void FindMinutia(CUDAArray<float> image, CUDAArray<float> orientationField, CUDAArray<bool> visited,
	CUDAArray<int> countOfMinutiae, CUDAArray<Minutiae> minutiaes, 
	const int size, const int step, int colorThreshold = 15)
{
	Point* section = new Point[size];
	float sectionAngle;
	int centerSection;
	bool flag;
	//minutiaes.SetAt(blockIdx.x * gridDim.x + blockIdx.y, 0, new ListOfMinutiae);

	int partX = 32; //image.Height / gridDim.x;
	int partY = 32; //image.Width / gridDim.y;

	int indexOfMinutiae = blockIdx.x * image.Height + blockIdx.y * defaultThreadCount;

	//printf("%d %d %d\n", blockIdx.x, blockIdx.y, threadIdx.x);

	//if (blockIdx.x == 11 && blockIdx.y == 7)
	for (int i = blockIdx.x * partX; i < (blockIdx.x + 1) * partX; i++)
		for (int j = blockIdx.y * partY; j < (blockIdx.y + 1) * partY; j++)
	//for (int i = 0; i < image.Height; i++)
	//	for (int j = 0; j < image.Width; j++)
		{
			if (OutOfImage(image, i, j, partX, partY))
			{
				//printf("Tu-tu. %d %d\n", i, j);
				continue;
			}

			if ((image.At(i, j) >= colorThreshold) || visited.At(i, j)) continue;
			visited.SetAt(i, j, true);
			//if (blockIdx.x == 0 && blockIdx.y == 0) printf("%d %d: Point (%d, %d) has been visited\n", blockIdx.x, blockIdx.y, i, j);

			//printf("%d %d:\n", i, j);

			FollowLine(i, j, Forward, image, orientationField, visited, countOfMinutiae, minutiaes, 
				section, &sectionAngle, &centerSection, &flag, size, step, partX, partY, &indexOfMinutiae);
			FollowLine(i, j, Back, image, orientationField, visited, countOfMinutiae, minutiaes, 
				section, &sectionAngle, &centerSection, &flag, size, step, partX, partY, &indexOfMinutiae);
		}
}

ListOfMinutiae* MergeMinutiaePools(ListOfMinutiae** pools)
{
	ListOfMinutiae* resPool = new ListOfMinutiae;

	for (int i = 0; i < defaultThreadCount; i++)
	{
		while (pools[i]->head != NULL)
		{
			resPool->Add(pools[i]->Pop());
		}
	}

	return resPool;
}

int CountOfMinutiaes(int* counts, int length)
{
	int count = 0;

	for (int i = 0; i < length; i++)
	{
		printf("%d ", counts[i]);
		count += counts[i];
	}

	return count;
}

bool* Start(float* source, int step, int lengthWings, int width, int height)
{
	int sizeSection = lengthWings * 2 + 1;

	CUDAArray<float> image = CUDAArray<float>(source, width, height);

	dim3 blockSize = 1;
	dim3 gridSize = dim3(ceilMod(image.Height, defaultThreadCount), ceilMod(image.Width, defaultThreadCount));

	CUDAArray<float> orientationField = CUDAArray<float>(OrientationFieldInBlocks(source, width, height), height, width);
	CUDAArray<bool> visited = CUDAArray<bool>((bool*)calloc(width * height, sizeof(bool)), width, height);
	CUDAArray<int> countOfMinutiae = CUDAArray<int>((int*)calloc(gridSize.x * gridSize.y, sizeof(int)), gridSize.x * gridSize.y, 1);
	CUDAArray<Minutiae> minutiaes = CUDAArray<Minutiae>((Minutiae*)calloc(width * height, sizeof(Minutiae)), width * height, 1);
	//CUDAArray<ListOfMinutiae*> minutiaes = CUDAArray<ListOfMinutiae*>((ListOfMinutiae**)calloc(gridSize.x * gridSize.y, sizeof(ListOfMinutiae*)), gridSize.x * gridSize.y, 1);

	FindMinutia << <gridSize, blockSize >> > (image, orientationField, visited, countOfMinutiae, minutiaes, sizeSection, step);
	hipDeviceSynchronize();
	hipError_t e = hipGetLastError(); 
	if (e != hipSuccess) {
		printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));
		exit(0);
	}

	int* counts = countOfMinutiae.GetData();

	for (int i = 0; i < gridSize.x * gridSize.y; i++)
	{
		printf("%d ", counts[i]);
	}

	//CountOfMinutiaes(countOfMinutiae.GetData(), gridSize.x * gridSize.y);

	return visited.GetData();

	/*ListOfMinutiae** notProcessedPools = minutiaes.GetData();

	return Parsing(MergeMinutiaePools(notProcessedPools));*/
}

int main(int argc, char *argv[])
{
	int width;
	int height;
	/*if (argc != 2)
	{
		printf("Need path to file");
		return 0;
	}*/
	char* filename = "H:\\GitHub\\CUDA-Fingerprinting\\Code\\CUDAFingerprinting.GPU.RidgeLine\\res.bmp";  //Write your way to bmp file
	int* img = loadBmp(filename, &width, &height);
	float* source = (float*)malloc(height*width*sizeof(float));
	for (int i = 0; i < height; i++)
		for (int j = 0; j < width; j++)
		{
			source[i * width + j] = (float)img[i * width + j];
		}

	bool* res = Start(source, 2, 3, width, height);
	
	for (int i = 0; i < height; i++)
		for (int j = 0; j < width; j++)
		{
			img[i * width + j] = res[i * width + j] ? 255 : 0;
		}



	saveBmp("..\\rez.bmp", img, width, height);

 	return 0;
}